#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "hip/hip_runtime.h"
#include "time.h"
#include "param.h"
#include <ctime>
#include "hipblas.h"

//namespace {
//    const float PI = 3.1415926;
//    const float M_2PI = PI * 2;
//    const float halfLength = 600;
//    float goal_pos[2] = { halfLength, 0 };
//    const float POINT_PASS_OFF = 200;
//}

// С������
#define ENEMY_NUM 6
#define SELF_NUM 6  // ������������Ŀ
#define POS_INFO_LENGTH 6 // ÿ��������λ����ռ��float��Ŀ
#define GPU_COEF_BLOCKSCORE -1
#define GPU_COEF_DISTSCORE 0
#define GPU_COEF_NEARSCORE 1.5
#define INPUT_DIM 10
#define HIDDEN_LAYER_DIM 80
#define OUTPUT_DIM 50
__constant__ float PI = 3.1415926;
__constant__ float M_2PI = 6.2831852;
__constant__ float PITCH_WIDTH = 600;
__constant__ float PITCH_LENGTH = 900;
__constant__ float PENALTY_WIDTH = 200;
__constant__ float PENALTY_DEPTH = 100;
__constant__ float POINT_PASS_OFF = 80.0;
// �󳡲���
//#define ENEMY_NUM 8
//#define SELF_NUM 8  // ������������Ŀ
//#define POS_INFO_LENGTH 6 // ÿ��������λ����ռ��float��Ŀ
//#define GPU_COEF_BLOCKSCORE -1
//#define GPU_COEF_DISTSCORE 0
//#define GPU_COEF_NEARSCORE 1.5
//#define INPUT_DIM 10
//#define HIDDEN_LAYER_DIM 80
//#define OUTPUT_DIM 50
//__constant__ float PI = 3.1415926;
//__constant__ float M_2PI = 6.2831852;
//__constant__ float PITCH_WIDTH = 900;
//__constant__ float PITCH_LENGTH = 1200;
//__constant__ float PENALTY_WIDTH = 280;
//__constant__ float PENALTY_DEPTH = 120;
//__constant__ float POINT_PASS_OFF = 80.0;

// ����ʹ��constant����ٶȣ�������û�е�ͨ
//__constant__ float GPU_A1_MATRIX[INPUT_DIM * HIDDEN_LAYER_DIM];
//__constant__ float GPU_BIAS1_MATRIX[HIDDEN_LAYER_DIM];
//__constant__ float GPU_A2_MATRIX[HIDDEN_LAYER_DIM * OUTPUT_DIM];
//__constant__ float GPU_BIAS2_MATRIX[OUTPUT_DIM];


extern "C" void get_gpu_info() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
//        std::cout << "ʹ��GPU device " << i << ": " << devProp.name << std::endl;
//        std::cout << "�豸ȫ���ڴ������� " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
//        std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
//        std::cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
//        std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
//        std::cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << devProp.regsPerBlock << std::endl;
//        std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
//        std::cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
//        std::cout << "�豸�϶ദ������������ " << devProp.multiProcessorCount << std::endl;
//        std::cout << "======================================================" << std::endl;
        std::cout << "GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    }
}

// �ж�һ�����Ƿ��ڽ���
inline __device__ bool is_in_penalty(float pos_x, float pos_y) {
    if (abs(pos_y) <= PENALTY_WIDTH / 2 && abs(pos_x) >= (PITCH_LENGTH / 2 - PENALTY_DEPTH)) {
        return true;
    }
    else {
        return false;
    }
}

// ���������ľ���
inline __device__ float dist(float* pos_ptr1, float* pos_ptr2) {
    return sqrt((pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]));
}

// ���������ľ���
inline __device__ float dist(float x1, float y1, float * pos_ptr2) {
    return sqrt((x1 - pos_ptr2[0]) * (x1 - pos_ptr2[0]) + (y1 - pos_ptr2[1]) * (y1 - pos_ptr2[1]));
}

// ������������ƽ��
inline __device__ float dist2(float* pos_ptr1, float* pos_ptr2) {
    return (pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]);
}

// �������㹹�������ķ���
inline __device__ float dir(float* pos_ptr1, float* pos_ptr2) {
    return std::atan2(pos_ptr2[1] - pos_ptr1[1], pos_ptr2[0] - pos_ptr1[0]);
}

// ֱ����ֲ��Normalize������ϸ��û�п���
inline __device__ float Normalize(float angle)
{
    // ���ٴֵ���
    angle -= (int)(angle / M_2PI) * M_2PI;
    // ϸ���� (-PI,PI]
    while (angle > PI) {
        angle -= M_2PI;
    }

    while (angle <= -PI) {
        angle += M_2PI;
    }
    return angle;
}

// line_statusΪ1��ʾֱ��б�ʲ�Ϊ������������ʱ��ֱ�߷��� y=ax+b
// line_statusΪ0��ʾֱ��б��Ϊ������������ʱ��ֱ�߷���Ϊx=a
__device__ int get_line(float* line_point1, float* line_point2, float& a, float& b) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        a = line_point1[0];
        b = 0;
        return 0;
    }
    else {
        // ֱ�߷���:y=ax+b
        a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        b = line_point1[1] - a * line_point1[0];
        return 1;
    }
}

// ��һ��������point��������������ֱ���ϵ�������
__device__ float point2line_dist(float *point, float *line_point1, float *line_point2) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        return abs(point[0] - line_point1[0]);
    }
    else {
        // ֱ�߷���:y=ax+b
        float a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        float b = line_point1[1] - a * line_point1[0];
        return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
    }
}

// ��һ��������point��������������ֱ�߷��̵�a��b��y=ax+b
__device__ float point2line_dist(float* point, float a, float b) {
    return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
}

// ����ͶӰ�㣨���㣩
__device__ void get_projection(float a, float b, int line_status, float* point, float * projection_point) {
    if (line_status) { // ֱ��б��Ϊ����ֵ
        if (abs(a) < 1e-5) { // ֱ����x��ƽ��
            projection_point[0] = point[0];
            projection_point[1] = b;
        }
        else { 
            projection_point[0] = (point[0] / a + point[1] - b) / (a + 1 / a);
            projection_point[1] = a * projection_point[0] + b;
        }
    }
    else { // ֱ��б��Ϊ����
        projection_point[0] = a;
        projection_point[1] = point[1];
    }
}

// ��һ�������볡����
__device__ void MakeInField(float* point) {
    if (point[0] < - PITCH_LENGTH / 2) point[0] = - PITCH_LENGTH / 2;
    if (point[0] > PITCH_LENGTH / 2) point[0] = PITCH_LENGTH / 2;
    if (point[1] < -PITCH_WIDTH / 2) point[1] = -PITCH_WIDTH / 2;
    if (point[1] > PITCH_WIDTH / 2) point[1] = PITCH_WIDTH / 2;
}

// ��һ�����Ƴ��з�����
__device__ void MakeOutOfTheirPenaltyArea(float* point) {
    if (point[1] > 0) {
        if (point[0] - PITCH_LENGTH / 2 + PENALTY_DEPTH < PENALTY_WIDTH / 2 - point[1]) {
            point[0] = PITCH_LENGTH / 2 - PENALTY_DEPTH;
        }
        else {
            point[1] = PENALTY_WIDTH / 2;
        }
    }
    else {
        if (point[0] - PITCH_LENGTH / 2 + PENALTY_DEPTH < PENALTY_WIDTH / 2 + point[1]) {
            point[0] = PITCH_LENGTH / 2 - PENALTY_DEPTH;
        }
        else {
            point[1] = -PENALTY_WIDTH / 2;
        }
    }
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
inline __device__ float evaluate_dist(float dist) {
    if (dist < 200) { // ���뿪���һ������֮��
        return 200 - dist + 100;  // 200 - dist
    }
    else if (dist > 500) {
        return (dist - 500) / 100;  // dist - 500
    }
    else {
        return 0;
    }
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_receive(float *me_pos, float *ball_pos, float * their_player_ptr) {
    float reverseReceiveP = 0.0;

    float a,b;
    int line_status;
    line_status = get_line(ball_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            // �򵽵з��ľ���
            float* their_palyer_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float dist_ball2opp = dist(ball_pos, their_palyer_pos);
            if (dist_ball2opp < 20) {
                continue;
            }
            //// �жϵз��Ƿ��������Լ��м�
            float r = ((their_palyer_pos[0]-me_pos[0]) * (ball_pos[0] - me_pos[0]) + (their_palyer_pos[1] - me_pos[1]) * (ball_pos[1] - me_pos[1])) / (dist2(me_pos, ball_pos));
            if (r < 0 || r > 1) {
                continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_palyer_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_palyer_pos);
            float ball2proj_dist = dist(projection_point, ball_pos);

            if (opp2proj_dist > 300 || ball2proj_dist < 10) { // �з����봫���߽�Զ��з�������Ͻ������Բ����ǵз��Դ����ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / ball2proj_dist;
            // ����з���Ա�ٶȽ������ٵ�0.6����ñ�ֵ����0.6ʱ�õз���Ա�Դ�����Ӱ��
            // TODO ����ԽԶ�����ֵӦ��Խ��
            if (ratio_dist < 0.6) { 
                reverseReceiveP += -100 * ratio_dist + 60;
            }
        }
    }

    // ��Ҫ�赲�Լ�������
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    float me2ball_dist = dist(me_pos, ball_pos);
    float blockMyself = point2line_dist(me_pos, ball_pos, goal_pos);
    if (blockMyself <= 50 && me2ball_dist < 400 && ball_pos[0] > 100) {
        reverseReceiveP += POINT_PASS_OFF;
    }
    else {
        reverseReceiveP += 20.0 / (blockMyself + 0.1);
    }

    // ���ܹ��ڿ������ߣ����ߴ����׽Ӳ�����
    if (abs(me_pos[1]) > PITCH_WIDTH / 2 - 100) {
        reverseReceiveP += (abs(me_pos[1]) - (PITCH_WIDTH / 2 - 100));
    }

    return reverseReceiveP;
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_goal(float* me_pos, float* ball_pos, float* their_player_ptr) {
    float reverseGoalP = 0.0;
    float a, b;
    int line_status;
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    line_status = get_line(goal_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            // ���ŵ��з��ľ���
            float* their_palyer_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float dist_goal2opp = dist(goal_pos, their_palyer_pos);
            if (dist_goal2opp < 60) {
                continue;
            }
            // �жϵз��Ƿ��������Լ��м�
            float r = ((their_palyer_pos[0] - me_pos[0]) * (ball_pos[0] - me_pos[0]) + (their_palyer_pos[1] - me_pos[1]) * (ball_pos[1] - me_pos[1])) / (dist2(me_pos, goal_pos));
            if (r < 0 || r > 1) {
                continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_palyer_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_palyer_pos);
            float ball2proj_dist = dist(projection_point, ball_pos);

            if (opp2proj_dist > 300 || ball2proj_dist < 10) { // �з����봫���߽�Զ��з�������Ͻ������Բ����ǵз��Դ����ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / ball2proj_dist;
            // ����ʱ���ٽϴ��������㷨ʹ�������е���м��㣬ʵ�����Ż��и���ѡ����˵��˶��Լ����ŵ��赲���С
            if (ratio_dist < 0.3) {
                reverseGoalP += -50 * ratio_dist + 15;
            }
        }
    }
//    // ���ڿ���y=0���߲��׽��������
//    if (abs(me_pos[1]) < 50) {
//        reverseGoalP += 10;
//    }
    return reverseGoalP;
}

__global__ void gpu_calc(float startPos[], float map[])
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // ��������Ĳ���
    int step = startPos[2];
    int me_x = blockIdx.x*step + startPos[0];
    int me_y = threadIdx.x*step + startPos[1];
    // Ϊ������ɫһһ��Ӧ�����ֵ��Ҫ����255
    if (is_in_penalty(me_x, me_y)) {
        map[i] = 255;
    }
    else {
        float me_pos_ptr[2] = { me_x, me_y };
        float* ball_pos_ptr = startPos + 3;
        float* ball_vel_ptr = startPos + 5;
        float* our_player_ptr = startPos + 7;
        float* their_player_ptr = startPos + (7 + POS_INFO_LENGTH * SELF_NUM);
        // ����
        // ��Ҫʹ�õ�һЩ����

        float me2ball_dist = dist(me_pos_ptr, ball_pos_ptr);

        // ����ֵ
        float dist_value = evaluate_dist(me2ball_dist);
        float receive_value = evaluate_receive(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        float goal_value = evaluate_goal(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        float total_value = dist_value + receive_value + goal_value;
        // float total_value = receive_value;
        // map[i] = me2ball_dist;
        if (total_value < 255) {
            map[i] = total_value;
        }
        else {
            map[i] = 255;
        }
    }
}

extern "C" void calc_with_gpu(float* map_cpu, float* start_pos_cpu, int height, int width, int pos_num, float *pitch_info) {
    //clock_t begin, end;
    //begin = clock();
    float* map_gpu, * start_pos_gpu;

    int map_size = height * width * sizeof(float);
    int pos_size = pos_num * sizeof(float);

    hipError_t  status1 = hipMalloc((void**)&map_gpu, map_size); // ���ڴ洢�����Ľ��
    hipError_t  status2 = hipMalloc((void**)&start_pos_gpu, pos_size);
    if (status1 != hipSuccess || status2 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("**************** map calc cuda malloc dbuf error ******************* \r\n");
        return;
    }

    // ��������
    hipError_t status_cpy = hipMemcpy(start_pos_gpu, start_pos_cpu, pos_size, hipMemcpyHostToDevice);
    if (status_cpy != hipSuccess)
    {
        printf("**************** map calc cuda cpy error ******************* \r\n");
        return;
    }

    // ��GPU�п���������Ϣ
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_LENGTH"), pitch_info, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_WIDTH"), pitch_info + 1, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_DEPTH"), pitch_info + 2, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_WIDTH"), pitch_info + 3, sizeof(float));


    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimGrid(height);
    dim3 dimBlock(width);

    // ִ��kernel
    gpu_calc << <dimGrid, dimBlock >> > (start_pos_gpu, map_gpu);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(map_cpu, map_gpu, map_size, hipMemcpyDeviceToHost);

    // �ͷ�GPU�˵��ڴ�
    hipFree(map_gpu);
    hipFree(start_pos_gpu);
    //end = clock();
    //std::cout << "best support point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}


////////////////////////////////////////////////
//       break point calculator from csy      //

// pos info����
// (2+1+2+1+2+2+OURPLAYER_NUM*_palyer_pos_num+THEIRPLAYER_NUM*_palyer_pos_num) * sizeof(float)
// ����Ϊ����������Բ�ġ�max_drbble_dist����ǰ��Աλ�á���Ա�������λ�á�����ٶȡ��ҷ�С����λ�á������ٶȣ���λΪ�Ƿ�valid�����з�С����λ�á������ٶȣ���λΪ�Ƿ�valid��
// ����޸��ⲿ�ִ��룬����ϸ�Ķ���ֵ��GPU���ִ��벢��֮������Ӧ���޸�
// ��Ҫע��Ĳ�����CPU�ռ�����룬������и�ֵ�����俽����GPU��ʱ����Ŀռ䡢GPU�Ը��б���Ϣ�Ľ���
__global__ void break_gpu_calc(float pos_info[], float target_info[], float results[], int angle_mod, int dist_mod, float vis_points[])
{
    extern __shared__  float score_map[];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int target_point_idx = blockIdx.x;
    int ang_dist_idx = threadIdx.x;
    int dist_idx = ang_dist_idx % (dist_mod - 1);
    int ang_idx = (ang_dist_idx - dist_idx) / (dist_mod - 1);
    
    // �Ӿ���Ϣ������
    float* dribble_center_point = pos_info;
    float max_dribble_dist = pos_info[2];
    float* self_pos = pos_info + 3;
    float* ball_pos = pos_info + 6;
    float* target_point = target_info + 2 * target_point_idx;
    float* our_player_ptr = pos_info + 10;
    float* their_player_ptr = pos_info + (10 + POS_INFO_LENGTH * SELF_NUM);

    float point_score = 10000;

    //float mod = max_dribble_dist * 2 / (dist_idx + 1);
    float mod = 2 * max_dribble_dist * (dist_idx + 1) / (dist_mod - 1);
    float angle = (ang_idx - angle_mod) * PI / angle_mod + dir(self_pos, target_point);// +(ang_idx - angle_mod) * PI / angle_mod;

    float test_point[2];
    test_point[0] = self_pos[0] + mod * cos(angle);
    test_point[1] = self_pos[1] + mod * sin(angle);
    
    // ʹ��������Բ��
    float vec_dist = dist(test_point, dribble_center_point);
    if (vec_dist > max_dribble_dist) {
        float boundary_point[2];
        boundary_point[0] = (test_point[0] - dribble_center_point[0]) / vec_dist * max_dribble_dist;
        boundary_point[1] = (test_point[1] - dribble_center_point[1]) / vec_dist * max_dribble_dist;
        test_point[0] = dribble_center_point[0] + boundary_point[0];
        test_point[1] = dribble_center_point[1] + boundary_point[1];
    }

    // ���ƶ���������
    MakeInField(test_point);

    // ���ƶ����з�����
    //MakeOutOfTheirPenaltyArea(test_point);

    bool temp_flag = false;
    float a, b;
    int line_status = get_line(self_pos, test_point, a, b);
    for (int i = 0; i < ENEMY_NUM; i++) {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float test2enemy_dist = dist(their_player_pos, test_point);
            if (test2enemy_dist < 20) {
                temp_flag = true;
                break;
            }
            //// �жϵз��Ƿ���test_point���Լ��м�
            float r = ((their_player_pos[0] - self_pos[0]) * (test_point[0] - self_pos[0]) + (their_player_pos[1] - self_pos[1]) * (test_point[1] - self_pos[1])) / (dist2(self_pos, test_point));
            if (r < 0 || r > 1) {
                continue;
            }
            float projection_point[2];
            get_projection(a, b, line_status, their_player_pos, projection_point);
            if (dist(projection_point, their_player_pos) < 40) {
                temp_flag = true;
                break;
            }
        }
    }

    int cnt = 0;
    if (temp_flag) {
        point_score = 9999;
    }
    else {
        float a1, b1;
        int line_status1 = get_line(test_point, target_point, a1, b1);
        
        // ����ָ��
        float dist_score = dist(test_point, target_point);
        
        float block_score = 8888, near_score = 9999;
        
        for (int i = 0; i < ENEMY_NUM; i++) {
            if (their_player_ptr[i * POS_INFO_LENGTH]) {
                float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
                float straight_dist = dist(their_player_pos, test_point);
                near_score = min(near_score, straight_dist);
                //// �жϵз��Ƿ���test_point��target_point�м�
                float r = ((their_player_pos[0] - target_point[0]) * (test_point[0] - target_point[0]) + (their_player_pos[1] - target_point[1]) * (test_point[1] - target_point[1])) / (dist2(target_point, test_point));
                if (r < 0 || r > 1) {
                    continue;
                }
                cnt++;
                float projection_point[2];
                get_projection(a1, b1, line_status1, their_player_pos, projection_point);
                float projection_dist = dist(projection_point, their_player_pos);
                if (projection_dist > 60) {
                    continue;
                }
                block_score = min(projection_dist, block_score);
            }
        }
        near_score = 1 / (near_score + 1e-8);
        point_score = GPU_COEF_BLOCKSCORE * block_score + GPU_COEF_DISTSCORE * dist_score + GPU_COEF_NEARSCORE * near_score;
    }

    score_map[3 * ang_dist_idx] = point_score;
    score_map[3 * ang_dist_idx + 1] = test_point[0];
    score_map[3 * ang_dist_idx + 2] = test_point[1];
    if (blockIdx.x == 0) {
        vis_points[3 * ang_dist_idx] = point_score;
        vis_points[3 * ang_dist_idx + 1] = test_point[0];
        vis_points[3 * ang_dist_idx + 2] = test_point[1];
    }
    __syncthreads();

    float best_score = 20000;
    int best_idx = 0;
    if (threadIdx.x == 0) {
        for (int i = 0; i < (angle_mod * 2 - 1) * (dist_mod - 1); i++) {
            if (score_map[3 * i] < best_score) {
                best_score = score_map[3 * i];
                best_idx = i;
            }
        }
        results[3 * blockIdx.x] = best_score;
        results[3 * blockIdx.x + 1] = score_map[3 * best_idx + 1];
        results[3 * blockIdx.x + 2] = score_map[3 * best_idx + 2];
    }
    //if (blockIdx.x == 0 && threadIdx.x == 0) {
    //    memcpy(vis_points, score_map, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float));
    //}
}


// target_point_cpu�������ϵĵ�
// target_point_num��Ŀ�����Ŀ
// pos_info_cpu���Ӿ���Ϣ����ͻ����˵�λ��
// pos_info_num���Ӿ���Ϣ��Ŀ

extern "C" int break_calc_with_gpu(float* target_point_cpu, int target_point_num, float* pos_info_cpu, int pos_info_num, int angle_mod, int dist_mod, float* results, float* vis_points_cpu) {
    //clock_t begin, end;
    //begin = clock();
    float* results_gpu, * pos_info_gpu, * target_point_gpu, * vis_points_gpu;

    int result_size = target_point_num * 3 * sizeof(float);
    int pos_size = pos_info_num * sizeof(float);
    int target_info_size = target_point_num * 2 * sizeof(float);

    //std::cout << "pos_info_cpu: " << std::endl;
    //for (int i = 0; i < pos_info_num; i++) {
    //    std::cout << pos_info_cpu[i] << " ";
    //}
    //std::cout << std::endl;
    // ��ȡ�ռ�
    hipError_t  status1 = hipMalloc((void**)&results_gpu, result_size); // ���ڴ洢�����Ľ��
    hipError_t  status2 = hipMalloc((void**)&pos_info_gpu, pos_size);
    hipError_t  status3 = hipMalloc((void**)&target_point_gpu, target_info_size);
    hipError_t  status4 = hipMalloc((void**)&vis_points_gpu, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float));
    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess)
    {
        printf("****************cuda malloc dbuf error ******************* \r\n");
        return 0;
    }

    // ��������
    hipError_t status_cpy1 = hipMemcpy(pos_info_gpu, pos_info_cpu, pos_size, hipMemcpyHostToDevice);
    hipError_t status_cpy2 = hipMemcpy(target_point_gpu, target_point_cpu, target_info_size, hipMemcpyHostToDevice);
    if (status_cpy1 != hipSuccess || status_cpy2 != hipSuccess)
    {
        printf("****************cuda cpy error ******************* \r\n");
        return 0;
    }

    // ��GPU�п���������Ϣ
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_LENGTH"), pitch_info, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_WIDTH"), pitch_info + 1, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_DEPTH"), pitch_info + 2, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_WIDTH"), pitch_info + 3, sizeof(float));


    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimGrid(target_point_num);
    dim3 dimBlock((angle_mod * 2 - 1) * (dist_mod - 1));

    // ִ��kernel
    break_gpu_calc << <dimGrid, dimBlock, 3 * (angle_mod * 2 - 1)* (dist_mod - 1) * sizeof(float) >> > (pos_info_gpu, target_point_gpu, results_gpu, angle_mod, dist_mod, vis_points_gpu); // ����������ָ�������ڴ��С

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(results, results_gpu, result_size, hipMemcpyDeviceToHost);
    hipMemcpy(vis_points_cpu, vis_points_gpu, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float), hipMemcpyDeviceToHost);
    // �ͷ�GPU�˵��ڴ�
    hipFree(target_point_gpu);
    hipFree(pos_info_gpu);
    hipFree(results_gpu);
    return 1;

    //end = clock();
    //std::cout << "best break point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}


// mat_c = mat_a * mat_b
//�Ż�������cublas��hipblasSgemm���پ�������
// mat_a:M*N, mat_b:N*K, mat_c:M*K
//__global__ void matrix_multi(float* mat_a, float* mat_b, float* mat_c, int M, int N, int K) {
void matrix_multi(float* mat_a, float* mat_b, float* mat_c, int M, int N, int K) {

    hipblasHandle_t handle;
    // initialize CUBLAS context
    hipblasStatus_t stat = hipblasCreate(&handle);

    float al = 1.0f;
    float bet = 0.0f;
    
    //��ǰ�϶�ԭ����洢��ʽΪ������
    stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, N, 
        &al, mat_a, N, mat_b, K, 
        &bet, mat_c, M);
    
    //��Ϊ�����ȿ��������д���
    //stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
    //    &al, mat_a, M, mat_b, K,
    //    &bet, mat_c, M);
    hipblasDestroy(handle);
    //int row_idx = blockIdx.x;
    //int col_idx = threadIdx.x;
    //int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //float c = 0;
    //for (int i = 0; i < N; i++) {
    //    c += mat_a[row_idx * N + i] * mat_b[col_idx * N + i];
    //}
    //mat_c[idx] = c;

}

__global__ void matrix_add(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    MatC[idx] = MatA[idx] + MatB[idx];
}

__global__ void matrix_add_with_relu(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    MatC[idx] = max(MatA[idx] + MatB[idx], 0.0f);
}

//// ��������Ԥ��Ĳ���
//extern "C" int set_ball_model_param(float* a_1_matrix_cpu, float* bias_1_matrix_cpu, float* a_2_matrix_cpu, float* bias_2_matrix_cpu) {
//
//    float* a_1_matrix_gpu, * bias_1_matrix_gpu, * a_2_matrix_gpu, * bias_2_matrix_gpu;
//
//    int result_size = OUTPUT_DIM * sizeof(float);
//    int vel_data_size = INPUT_DIM * sizeof(float);
//    int hidden_layer_size = HIDDEN_LAYER_DIM * sizeof(float);
//
//    hipError_t status1 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_A1_MATRIX), a_1_matrix_cpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float));
//    hipError_t status2 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_BIAS1_MATRIX), bias_1_matrix_cpu, HIDDEN_LAYER_DIM * sizeof(float));
//    hipError_t status3 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_A2_MATRIX), a_2_matrix_cpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float));
//    hipError_t status4 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_BIAS2_MATRIX), bias_2_matrix_cpu, OUTPUT_DIM * sizeof(float));
//
//    //for (int i = 0; i < 10; i++) {
//    //    std::cout << bias_1_matrix_cpu[i] << " ";
//    //}
//    //std::cout << std::endl << std::endl;
//
//    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess || status4 != hipSuccess)
//    {
//        printf("**************** set ball model cuda malloc dbuf error ******************* \r\n");
//        return 0;
//    }
//    return 1;
//}

// ����Ԥ�ⲿ�֣���������һЩGPU�ϵľ�������
extern "C" void ball_model_calc_with_gpu(float* vel_data_cpu, float *predict_results, float* a_1_matrix_cpu, float* bias_1_matrix_cpu, float* a_2_matrix_cpu, float* bias_2_matrix_cpu) {
    //clock_t begin, end;
    //begin = clock();
    float* results_gpu, * vel_data_gpu, * hidden_layer_data_gpu;
    float* a_1_matrix_gpu, * bias_1_matrix_gpu, * a_2_matrix_gpu, * bias_2_matrix_gpu;

    int result_size = OUTPUT_DIM * sizeof(float);
    int vel_data_size = INPUT_DIM * sizeof(float);
    int hidden_layer_size = HIDDEN_LAYER_DIM * sizeof(float);

    // ��ȡ�ռ�
    hipError_t status1 = hipMalloc((void**)&results_gpu, result_size); // ���ڴ洢�����Ľ��
    hipError_t status2 = hipMalloc((void**)&vel_data_gpu, vel_data_size);
    hipError_t status3 = hipMalloc((void**)&hidden_layer_data_gpu, hidden_layer_size);

    hipError_t status4 = hipMalloc((void**)&a_1_matrix_gpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float));
    hipError_t status5 = hipMalloc((void**)&bias_1_matrix_gpu, HIDDEN_LAYER_DIM * sizeof(float));
    hipError_t status6 = hipMalloc((void**)&a_2_matrix_gpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float));
    hipError_t status7 = hipMalloc((void**)&bias_2_matrix_gpu, OUTPUT_DIM * sizeof(float));
    

    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("    Eror text:  %s\n", hipGetErrorString(status1));
        printf("    Eror text:  %s\n", hipGetErrorString(status2));
        printf("    Eror text:  %s\n", hipGetErrorString(status3));
        printf("**************** ball model cuda malloc dbuf error ******************* \r\n");
        return;
    }
    if (status4 != hipSuccess || status5 != hipSuccess || status6 != hipSuccess || status7 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("    Eror text:  %s\n", hipGetErrorString(status4));
        printf("    Eror text:  %s\n", hipGetErrorString(status5));
        printf("    Eror text:  %s\n", hipGetErrorString(status6));
        printf("    Eror text:  %s\n", hipGetErrorString(status7));
        printf("**************** ball model cuda param malloc dbuf error ******************* \r\n");
        return;
    }

    // ��������
    hipError_t status_cpy1 = hipMemcpy(vel_data_gpu, vel_data_cpu, vel_data_size, hipMemcpyHostToDevice);
    hipError_t status_cpy2 = hipMemcpy(a_1_matrix_gpu, a_1_matrix_cpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy3 = hipMemcpy(bias_1_matrix_gpu, bias_1_matrix_cpu, HIDDEN_LAYER_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy4 = hipMemcpy(a_2_matrix_gpu, a_2_matrix_cpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy5 = hipMemcpy(bias_2_matrix_gpu, bias_2_matrix_cpu, OUTPUT_DIM * sizeof(float), hipMemcpyHostToDevice);
    if (status_cpy1 != hipSuccess || status_cpy2 != hipSuccess || status_cpy3 != hipSuccess || status_cpy4 != hipSuccess || status_cpy5 != hipSuccess)
    {
        printf("**************** ball model cuda cpy error ******************* \r\n");
        return;
    }

    // y=xA
    //dim3 dimGrid1(1);
    //dim3 dimBlock1(HIDDEN_LAYER_DIM);
    //matrix_multi << <dimGrid1, dimBlock1 >> > (vel_data_gpu, a_1_matrix_gpu, hidden_layer_data_gpu, 1, INPUT_DIM, HIDDEN_LAYER_DIM);
    matrix_multi(vel_data_gpu, a_1_matrix_gpu, hidden_layer_data_gpu, 1, INPUT_DIM, HIDDEN_LAYER_DIM);

    // relu(y+=b)���ӷ�����ʹ��ͬһƬ��ַ�����ǳ˷�����
    dim3 dimGrid2(1);
    dim3 dimBlock2(HIDDEN_LAYER_DIM);
    matrix_add_with_relu << <dimGrid2, dimBlock2 >> > (hidden_layer_data_gpu, bias_1_matrix_gpu, hidden_layer_data_gpu, 1, HIDDEN_LAYER_DIM);

    // y=xA
    //dim3 dimGrid3(1);
    //dim3 dimBlock3(OUTPUT_DIM);
    //matrix_multi << <dimGrid3, dimBlock3 >> > (hidden_layer_data_gpu, a_2_matrix_gpu, results_gpu, 1, HIDDEN_LAYER_DIM, OUTPUT_DIM);
    matrix_multi(hidden_layer_data_gpu, a_2_matrix_gpu, results_gpu, 1, HIDDEN_LAYER_DIM, OUTPUT_DIM);

    // y+=b���ӷ�����ʹ��ͬһƬ��ַ�����ǳ˷�����
    dim3 dimGrid4(1);
    dim3 dimBlock4(OUTPUT_DIM);
    matrix_add << <dimGrid4, dimBlock4 >> > (results_gpu, bias_2_matrix_gpu, results_gpu, 1, OUTPUT_DIM);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(predict_results, results_gpu, result_size, hipMemcpyDeviceToHost);

    // �ͷ�GPU�˵��ڴ�
    hipFree(results_gpu);
    hipFree(vel_data_gpu);
    hipFree(hidden_layer_data_gpu);
    hipFree(a_1_matrix_gpu);
    hipFree(bias_1_matrix_gpu);
    hipFree(a_2_matrix_gpu);
    hipFree(bias_2_matrix_gpu);
    //end = clock();
    //std::cout << "best break point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}