#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "hip/hip_runtime.h"
#include "time.h"
#include "param.h"
#include <ctime>
#include "hipblas.h"

//namespace {
//    const float PI = 3.1415926;
//    const float M_2PI = PI * 2;
//    const float halfLength = 600;
//    float goal_pos[2] = { halfLength, 0 };
//    const float POINT_PASS_OFF = 200;
//}

// С������
#define ENEMY_NUM 6
#define SELF_NUM 6  // ������������Ŀ
#define POS_INFO_LENGTH 6 // ÿ��������λ����ռ��float��Ŀ
#define GPU_COEF_BLOCKSCORE -1
#define GPU_COEF_DISTSCORE 0
#define GPU_COEF_NEARSCORE 1.5
#define INPUT_DIM 10
#define HIDDEN_LAYER_DIM 80
#define OUTPUT_DIM 50
__constant__ float MAX_SCORE = 255; // ����ֵ������ɫ��Ӧ
__constant__ float PI = 3.1415926;
__constant__ float G = 980.0;
__constant__ float M_2PI = 6.2831852;
__constant__ float PITCH_WIDTH = 600;
__constant__ float PITCH_LENGTH = 900;
__constant__ float PENALTY_WIDTH = 200;
__constant__ float PENALTY_DEPTH = 100;
__constant__ float GOAL_WIDTH = 100;
__constant__ float LINE_WIDTH = 1;
__constant__ float POINT_PASS_OFF = 80.0;
// �����˺�����������
__constant__ float BALL_RADIUS = 2.15;
__constant__ float BALL_DEC = 98.0; // ����������98.0
__constant__ float BALL_MAX_VEL = 650.0;
__constant__ float ROBOT_RADIUS = 9.0;
__constant__ float ROBOT_FRONT_TO_CENTER = 7.6;
__constant__ float ROBOT_MAX_HEIGHT = 15.0;
__constant__ float ROBOT_CHIP_ANGLE = 45.0;
__constant__ float ROBOT_MAX_ACC = 200.0; // �����ǳ�����������һЩ
__constant__ float ROBOT_MAX_DEC = 200.0;
__constant__ float ROBOT_MAX_VEL = 200.0;
__constant__ float ROBOT_MAX_ROTACC = 5.0;
__constant__ float ROBOT_MAX_ROTDEC = 5.0;
__constant__ float ROBOT_MAX_ROTVEL = 10.0;
// ���������ֵ
__constant__ float BEST_TIME_RATIO = 0.333;
__constant__ float WOREST_TIME_RATIO = 1.0;
__constant__ float FASTEST_RECEIVE_VEL = 80.0; // �ܽ�ס���������
__constant__ float LARGEST_GOAL_ANGLE = 80.0; // �����żн�̫�󲻴���
__constant__ float SHORTEST_PASS_DIST = 100.0; // ����̫��������
__constant__ float CHIP_TIME_FACTOR = 1.5; // �ӳ���ķ���ʱ�䣬Ӱ�������Ƿ����ص��ж�
__constant__ float LARGEST_TOUCH_ANGLE = 80.0;
// �󳡲���
//#define ENEMY_NUM 8
//#define SELF_NUM 8  // ������������Ŀ
//#define POS_INFO_LENGTH 6 // ÿ��������λ����ռ��float��Ŀ
//#define GPU_COEF_BLOCKSCORE -1
//#define GPU_COEF_DISTSCORE 0
//#define GPU_COEF_NEARSCORE 1.5
//#define INPUT_DIM 10
//#define HIDDEN_LAYER_DIM 80
//#define OUTPUT_DIM 50
//__constant__ float PI = 3.1415926;
//__constant__ float M_2PI = 6.2831852;
//__constant__ float PITCH_WIDTH = 900;
//__constant__ float PITCH_LENGTH = 1200;
//__constant__ float PENALTY_WIDTH = 280;
//__constant__ float PENALTY_DEPTH = 120;
//__constant__ float POINT_PASS_OFF = 80.0;

// ����ʹ��constant����ٶȣ�������û�е�ͨ
//__constant__ float GPU_A1_MATRIX[INPUT_DIM * HIDDEN_LAYER_DIM];
//__constant__ float GPU_BIAS1_MATRIX[HIDDEN_LAYER_DIM];
//__constant__ float GPU_A2_MATRIX[HIDDEN_LAYER_DIM * OUTPUT_DIM];
//__constant__ float GPU_BIAS2_MATRIX[OUTPUT_DIM];


extern "C" void get_gpu_info() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
//        std::cout << "ʹ��GPU device " << i << ": " << devProp.name << std::endl;
//        std::cout << "�豸ȫ���ڴ������� " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
//        std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
//        std::cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
//        std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
//        std::cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << devProp.regsPerBlock << std::endl;
//        std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
//        std::cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
//        std::cout << "�豸�϶ദ������������ " << devProp.multiProcessorCount << std::endl;
//        std::cout << "======================================================" << std::endl;
        std::cout << "GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    }
}

inline __device__ float pow(float a, int n) {
    float result = 1.0;
    if (n < 0 && a != 0)
        a = 1.0 / a;
    for (int i = 0; i < n; i++)
        result *= a;
    return result;
}

// �ж�һ�����Ƿ��ڽ���
inline __device__ bool is_in_penalty(float pos_x, float pos_y, float buffer) {
    if (abs(pos_y) <= PENALTY_WIDTH / 2 + buffer && abs(pos_x) >= (PITCH_LENGTH / 2 - PENALTY_DEPTH - buffer)) {
        return true;
    }
    else {
        return false;
    }
}

// ���������ľ���
inline __device__ float dist(float* pos_ptr1, float* pos_ptr2) {
    return sqrt((pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]));
}

// ���������ľ���
inline __device__ float dist(float x1, float y1, float * pos_ptr2) {
    return sqrt((x1 - pos_ptr2[0]) * (x1 - pos_ptr2[0]) + (y1 - pos_ptr2[1]) * (y1 - pos_ptr2[1]));
}

// ������������ƽ��
inline __device__ float dist2(float* pos_ptr1, float* pos_ptr2) {
    return (pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]);
}

// �������㹹�������ķ���
inline __device__ float dir(float* pos_ptr1, float* pos_ptr2) {
    return std::atan2(pos_ptr2[1] - pos_ptr1[1], pos_ptr2[0] - pos_ptr1[0]);
}

// ֱ����ֲ��Normalize������ϸ��û�п���
inline __device__ float Normalize(float angle)
{
    // ���ٴֵ���
    angle -= (int)(angle / M_2PI) * M_2PI;
    // ϸ���� (-PI,PI]
    while (angle > PI) {
        angle -= M_2PI;
    }

    while (angle <= -PI) {
        angle += M_2PI;
    }
    return angle;
}

// �Ƕ�ֵתΪ������
inline __device__ float d2r(float degree)
{
    float rad = degree * PI / 180.0;
    rad = Normalize(rad);
    return rad;
}

// line_statusΪ1��ʾֱ��б�ʲ�Ϊ������������ʱ��ֱ�߷��� y=ax+b
// line_statusΪ0��ʾֱ��б��Ϊ������������ʱ��ֱ�߷���Ϊx=a
__device__ int get_line(float* line_point1, float* line_point2, float& a, float& b) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        a = line_point1[0];
        b = 0;
        return 0;
    }
    else {
        // ֱ�߷���:y=ax+b
        a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        b = line_point1[1] - a * line_point1[0];
        return 1;
    }
}

// ��һ��������point��������������ֱ���ϵ�������
__device__ float point2line_dist(float *point, float *line_point1, float *line_point2) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        return abs(point[0] - line_point1[0]);
    }
    else {
        // ֱ�߷���:y=ax+b
        float a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        float b = line_point1[1] - a * line_point1[0];
        return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
    }
}

// ��һ��������point��������������ֱ�߷��̵�a��b��y=ax+b
__device__ float point2line_dist(float* point, float a, float b) {
    return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
}

// ����ͶӰ�㣨���㣩
__device__ void get_projection(float a, float b, int line_status, float* point, float * projection_point) {
    if (line_status) { // ֱ��б��Ϊ����ֵ
        if (abs(a) < 1e-5) { // ֱ����x��ƽ��
            projection_point[0] = point[0];
            projection_point[1] = b;
        }
        else { 
            projection_point[0] = (point[0] / a + point[1] - b) / (a + 1 / a);
            projection_point[1] = a * projection_point[0] + b;
        }
    }
    else { // ֱ��б��Ϊ����
        projection_point[0] = a;
        projection_point[1] = point[1];
    }
}

// ��һ�������볡����
__device__ void MakeInField(float* point) {
    if (point[0] < - PITCH_LENGTH / 2) point[0] = - PITCH_LENGTH / 2;
    if (point[0] > PITCH_LENGTH / 2) point[0] = PITCH_LENGTH / 2;
    if (point[1] < -PITCH_WIDTH / 2) point[1] = -PITCH_WIDTH / 2;
    if (point[1] > PITCH_WIDTH / 2) point[1] = PITCH_WIDTH / 2;
}

// ��һ�����Ƴ��з�����
__device__ void MakeOutOfTheirPenaltyArea(float* point) {
    if (point[1] > 0) {
        if (point[0] - PITCH_LENGTH / 2 + PENALTY_DEPTH < PENALTY_WIDTH / 2 - point[1]) {
            point[0] = PITCH_LENGTH / 2 - PENALTY_DEPTH;
        }
        else {
            point[1] = PENALTY_WIDTH / 2;
        }
    }
    else {
        if (point[0] - PITCH_LENGTH / 2 + PENALTY_DEPTH < PENALTY_WIDTH / 2 + point[1]) {
            point[0] = PITCH_LENGTH / 2 - PENALTY_DEPTH;
        }
        else {
            point[1] = -PENALTY_WIDTH / 2;
        }
    }
}

// �����ٶȹ滮���������ٵ���ʱ�䣬dist����Ϊ����
inline __device__ float TrapezoidalMotionTime(float dist, int mode) {
    float max_acc;
    float max_dec;
    float max_vel;
    if (mode == 0) {
        max_acc = ROBOT_MAX_ACC;
        max_dec = ROBOT_MAX_DEC;
        max_vel = ROBOT_MAX_VEL;
    }
    else if (mode == 1) {
        max_acc = ROBOT_MAX_ROTACC;
        max_dec = ROBOT_MAX_ROTDEC;
        max_vel = ROBOT_MAX_ROTVEL;
    }

    float trapezoidal_motion_time = 0.0;
    // ������ٶ�Ϊ0
    float vel2max_dist = max_vel * max_vel / (2 * max_acc);
    if (vel2max_dist < dist) { // �ܼ��ٵ�����ٶ�
        float more_dist = dist - vel2max_dist;
        float vel2min_dist = max_vel * max_vel / (2 * max_dec);
        if (vel2min_dist <= more_dist) { // ���ڵ���ǰ�������٣��������ٽ׶�
            more_dist -= vel2min_dist;
            trapezoidal_motion_time = (max_vel / max_acc) + (max_vel / max_dec) + more_dist / max_vel;
        }
    }
    if (trapezoidal_motion_time == 0.0 && dist > 0) {
        // ���ܼ��ٵ�����ٶ�
        float max_vel = sqrt(2 * max_acc * max_dec * dist / (max_acc + max_dec));
        trapezoidal_motion_time = (max_vel / max_acc) + (max_vel / max_dec);
    }

    return trapezoidal_motion_time;
}

// �ж�һ�����Ƿ���ײ�Է�����������
inline __device__ bool if_collide_theirPlayer(float* pos, float* their_player_ptr) {
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            // ��λ���з��ľ���
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            if (dist(their_player_pos, pos) < 2 * ROBOT_RADIUS)
                return true;
        }
    }
    return false;
}

// �ж϶Է����Ž�
inline __device__ int check_their_goalie(float* their_player_ptr) {
    int their_goalie = 0;
    float min_dist = 9999;
    float goal[2] = { PITCH_LENGTH / 2, 0 };
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            if (is_in_penalty(their_player_pos[0], their_player_pos[1], 0)) {
                float temp_min_dist = dist(their_player_pos, goal);
                if (temp_min_dist < min_dist) {
                    min_dist = temp_min_dist;
                    their_goalie = i;
                }
            }
        }
    }
    return their_goalie;
}

// �ж϶Է����Ž�
inline __device__ int rolematch(float* me_pos, float* our_player_ptr) {
    int our_match_player = 0;
    // ��һ��RoleMatch���
    float min_dist = 9999;
    float match_player_pos[2];
    for (int j = 0; j < SELF_NUM; j++) {
        if (our_player_ptr[j * POS_INFO_LENGTH]) {
            float* our_player_pos = our_player_ptr + j * POS_INFO_LENGTH + 1;
            float temp_dist = dist(our_player_pos, me_pos);
            if (temp_dist < min_dist) {
                min_dist = temp_dist;
                our_match_player = j;
            }
        }
    }
    return our_match_player;
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
inline __device__ float evaluate_dist(float dist) {
    if (dist < 100) { // ���뿪���һ������֮��
        return 100 - dist + 100;  // 200 - dist
    }
    else if (dist > 500) {
        return (dist - 500) / 100;  // dist - 500
    }
    else {
        return 0;
    }
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_receive(float *me_pos, float *ball_pos, float * their_player_ptr) {
    float reverseReceiveP = 0.0;

    float a,b;
    int line_status;
    line_status = get_line(ball_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            // �򵽵з��ľ���
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float dist_ball2opp = dist(ball_pos, their_player_pos);
            if (dist_ball2opp < 20) {
                continue;
            }
            //// �жϵз��Ƿ��������Լ��м�
            float r = ((their_player_pos[0]-me_pos[0]) * (ball_pos[0] - me_pos[0]) + (their_player_pos[1] - me_pos[1]) * (ball_pos[1] - me_pos[1])) / (dist2(me_pos, ball_pos));
            if (r < 0 || r > 1) {
                continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_player_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_player_pos);
            float ball2proj_dist = dist(projection_point, ball_pos);           

            if (opp2proj_dist > 300 || ball2proj_dist < 10) { // �з����봫���߽�Զ��з�������Ͻ������Բ����ǵз��Դ����ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / ball2proj_dist;
            // ����з���Ա�ٶȽ������ٵ�0.6����ñ�ֵ����0.6ʱ�õз���Ա�Դ�����Ӱ��
            // TODO ����ԽԶ�����ֵӦ��Խ��
            if (ratio_dist < 0.6) { 
                reverseReceiveP += -100 * ratio_dist + 60;
            }
        }
    }

    // ��Ҫ�赲�Լ�������
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    float me2ball_dist = dist(me_pos, ball_pos);
    float blockMyself = point2line_dist(me_pos, ball_pos, goal_pos);
    if (blockMyself <= 50 && me2ball_dist < 400 && ball_pos[0] > 100) {
        reverseReceiveP += POINT_PASS_OFF;
    }
    else {
        reverseReceiveP += 20.0 / (blockMyself + 0.1);
    }
    
    // ���ܹ��ڿ������ߣ����ߴ����׽Ӳ�����
    if (abs(me_pos[0]) > PITCH_WIDTH / 2 - 20) { // 100
        reverseReceiveP += (abs(me_pos[0]) - (PITCH_WIDTH / 2 - 20));
    }

    return reverseReceiveP;
}

__device__ float evaluate_flat_pass_their(float* me_pos, float* ball_pos, float* their_player_ptr, int their_goalie) {
    float reverseReceiveP = 0.0;
    float a, b;
    int line_status;
    float ball_max_dist = BALL_MAX_VEL * BALL_MAX_VEL / (2 * BALL_DEC); // �����ɹ�����������
    float ball2me_dist = dist(ball_pos, me_pos); // �򵽵�λ�ľ���
    if (ball2me_dist - ROBOT_FRONT_TO_CENTER - BALL_RADIUS > ball_max_dist || ball2me_dist < SHORTEST_PASS_DIST) { // ̫Զ������ȥ����̫��û��Ҫ��
        reverseReceiveP = MAX_SCORE;
    }
    else {
        float ball_kicked_vel = sqrt(FASTEST_RECEIVE_VEL * FASTEST_RECEIVE_VEL + 2 * BALL_DEC * ball2me_dist); // ����ĳ�ʼ�ٶȣ�������
        ball_kicked_vel = min(ball_kicked_vel, BALL_MAX_VEL);
        line_status = get_line(ball_pos, me_pos, a, b);
        for (int j = 0; j < ENEMY_NUM; j++) {
            if (their_player_ptr[j * POS_INFO_LENGTH]) {
                float* their_player_pos = their_player_ptr + j * POS_INFO_LENGTH + 1;
                // ����ͶӰ��
                float projection_point[2];
                float intercept_point[2];
                get_projection(a, b, line_status, their_player_pos, projection_point);
                // ͶӰ���Ƿ�����͵�λ֮��
                if ((projection_point[0] - ball_pos[0]) * (projection_point[0] - me_pos[0]) + (projection_point[1] - ball_pos[1]) * (projection_point[1] - me_pos[1]) > 0) {
                    intercept_point[0] = me_pos[0];
                    intercept_point[1] = me_pos[1];
                }
                else {
                    bool in_penalty = is_in_penalty(projection_point[0], projection_point[1], 0);
                    if (j != their_goalie && in_penalty) { // ������Ա���ܽ�����
                        intercept_point[0] = me_pos[0];
                        intercept_point[1] = me_pos[1];
                    }
                    else if (j == their_goalie && !in_penalty) // ����Ա��������
                        continue;
                    else {
                        intercept_point[0] = projection_point[0];
                        intercept_point[1] = projection_point[1];
                    }                    
                }
                // ��պù��з�������ʱ��
                float ball2inter_dist = dist(intercept_point, ball_pos) + BALL_RADIUS;
                float ball2inter_vel = sqrt(ball_kicked_vel * ball_kicked_vel - 2 * BALL_DEC * ball2inter_dist);
                float ball2inter_time = (ball_kicked_vel - ball2inter_vel) / BALL_DEC;
                // �з���������պýӴ���������ʱ��
                float opp2inter_dist = dist(intercept_point, their_player_pos) - ROBOT_RADIUS;
                if (opp2inter_dist < 0)
                    opp2inter_dist = 0;
                float opp2inter_time = TrapezoidalMotionTime(opp2inter_dist, 0);

                float temp_score = 0.0;
                if (opp2inter_time == 0)
                    temp_score = MAX_SCORE;
                else {
                    float score_factor = min(ball2inter_time / opp2inter_time, 1.0);
                    temp_score = MAX_SCORE * powf(score_factor, 3);
                }
                if (reverseReceiveP < temp_score) // ȡ���ķ�ֵ���������ױ����صĵط�
                    reverseReceiveP = temp_score;
            }
        }
    }

    return reverseReceiveP;
}

__device__ float evaluate_flat_pass_our(float* me_pos, float* ball_pos, float* our_player_ptr) {
    float reverseReceiveP = 0.0;
    float ball_max_dist = BALL_MAX_VEL * BALL_MAX_VEL / (2 * BALL_DEC); // �����ɹ�����������
    float ball2me_dist = dist(ball_pos, me_pos); // �򵽵�λ�ľ���
    if (ball2me_dist - ROBOT_FRONT_TO_CENTER - BALL_RADIUS > ball_max_dist || ball2me_dist < SHORTEST_PASS_DIST) { // ̫Զ������ȥ����̫��û��Ҫ��
        reverseReceiveP = MAX_SCORE;
    }
    else {
        float ball_kicked_vel = sqrt(FASTEST_RECEIVE_VEL * FASTEST_RECEIVE_VEL + 2 * BALL_DEC * ball2me_dist); // ����ĳ�ʼ�ٶȣ�������
        ball_kicked_vel = min(ball_kicked_vel, BALL_MAX_VEL);
        float intercept_point[2];
        intercept_point[0] = me_pos[0];
        intercept_point[1] = me_pos[1];
        // ��պù��з�������ʱ��
        float ball2inter_dist = dist(intercept_point, ball_pos) + BALL_RADIUS;
        float ball2inter_vel = sqrt(ball_kicked_vel * ball_kicked_vel - 2 * BALL_DEC * ball2inter_dist);
        float ball2inter_time = (ball_kicked_vel - ball2inter_vel) / BALL_DEC;
        
        // �ҷ���������պýӴ���������ʱ��
        float our2inter_dist = dist(intercept_point, our_player_ptr) - ROBOT_RADIUS;
        if (our2inter_dist < 0)
            our2inter_dist = 0;
        float our2inter_time = TrapezoidalMotionTime(our2inter_dist, 0);

        if (ball2inter_time == 0)
            reverseReceiveP = MAX_SCORE;
        else {
            float score_factor = min(our2inter_time / ball2inter_time, 1.0);
            reverseReceiveP = MAX_SCORE * powf(score_factor, 3);
        }
    }

    return reverseReceiveP;
}

__device__ float evaluate_chip_pass_their(float* me_pos, float* ball_pos, float* their_player_ptr, int their_goalie) {
    float reverseReceiveP = 0.0;
    float a, b;
    int line_status;
    float chip_angle = d2r(ROBOT_CHIP_ANGLE);
    float ball_max_dist = BALL_MAX_VEL * cos(chip_angle) * BALL_MAX_VEL * sin(chip_angle) / G * 2; // �����ɷ��е�������
    float ball2me_dist = dist(ball_pos, me_pos) - ROBOT_FRONT_TO_CENTER - BALL_RADIUS; // �򵽵�λ�ľ��룬����������
    if (ball2me_dist > ball_max_dist || ball2me_dist < SHORTEST_PASS_DIST) { // ̫Զ������ȥ����̫��û��Ҫ��
        reverseReceiveP = MAX_SCORE;
    }
    else {
        float ball_kicked_vel = sqrt(G * ball2me_dist / (2 * sin(chip_angle) * cos(chip_angle))); // ����ĳ�ʼ�ٶ�
        line_status = get_line(ball_pos, me_pos, a, b);
        for (int j = 0; j < ENEMY_NUM; j++) {
            if (their_player_ptr[j * POS_INFO_LENGTH]) {
                float* their_player_pos = their_player_ptr + j * POS_INFO_LENGTH + 1;
                // ����ͶӰ��
                float projection_point[2];
                get_projection(a, b, line_status, their_player_pos, projection_point);
                // ͶӰ���Ƿ�����͵�λ֮��
                if ((projection_point[0] - ball_pos[0]) * (projection_point[0] - me_pos[0]) + (projection_point[1] - ball_pos[1]) * (projection_point[1] - me_pos[1]) < 0) {
                    if (dist(their_player_pos, ball_pos) - ROBOT_RADIUS < (ROBOT_MAX_HEIGHT + BALL_RADIUS) / tan(chip_angle)) { // ������̫��������ȥ
                        reverseReceiveP = MAX_SCORE;
                        break;
                    }
                }
                if (j == their_goalie) // ����Ա��������
                    continue;
                // �������
                float intercept_point[2];
                intercept_point[0] = me_pos[0];
                intercept_point[1] = me_pos[1];
                // ��պ���ص�ʱ�䣬�ʵ���ʱ���ǿ�������          
                float ball2inter_time = ball_kicked_vel * sin(chip_angle) / G * 2 * CHIP_TIME_FACTOR;
                // �з���������պýӴ���������ʱ��
                float opp2inter_dist = dist(intercept_point, their_player_pos) - ROBOT_RADIUS;
                if (opp2inter_dist < 0)
                    opp2inter_dist = 0;
                float opp2inter_time = TrapezoidalMotionTime(opp2inter_dist, 0);

                float temp_score = 0.0;
                if (opp2inter_time == 0)
                    temp_score = MAX_SCORE;
                else {
                    float score_factor = min(ball2inter_time / opp2inter_time, 1.0);
                    temp_score = MAX_SCORE * powf(score_factor, 3);
                }
                if (reverseReceiveP < temp_score) // ȡ���ķ�ֵ���������ױ����صĵط�
                    reverseReceiveP = temp_score;
            }
        }
    }

    return reverseReceiveP;
}

__device__ float evaluate_chip_pass_our(float* me_pos, float* ball_pos, float* our_player_ptr) {
    float reverseReceiveP = 0.0;
    float chip_angle = d2r(ROBOT_CHIP_ANGLE);
    float ball_max_dist = BALL_MAX_VEL * cos(chip_angle) * BALL_MAX_VEL * sin(chip_angle) / G * 2; // �����ɷ��е�������
    float ball2me_dist = dist(ball_pos, me_pos) - ROBOT_FRONT_TO_CENTER - BALL_RADIUS; // �򵽵�λ�ľ��룬����������
    if (ball2me_dist > ball_max_dist || ball2me_dist < SHORTEST_PASS_DIST) { // ̫Զ������ȥ����̫��û��Ҫ��
        reverseReceiveP = MAX_SCORE;
    }
    else {
        float ball_kicked_vel = sqrt(G * ball2me_dist / (2 * sin(chip_angle) * cos(chip_angle))); // ����ĳ�ʼ�ٶ�
        float intercept_point[2];
        intercept_point[0] = me_pos[0];
        intercept_point[1] = me_pos[1];
        // ��պ���ص�ʱ�䣬�ʵ���ʱ���ǿ�������
        float ball2inter_time = ball_kicked_vel * sin(chip_angle) / G * 2 * CHIP_TIME_FACTOR;
        // �ҷ���������պýӴ���������ʱ��
        float our2inter_dist = dist(intercept_point, our_player_ptr) - ROBOT_RADIUS;
        if (our2inter_dist < 0)
            our2inter_dist = 0;
        float our2inter_time = TrapezoidalMotionTime(our2inter_dist, 0);

        if (ball2inter_time == 0)
            reverseReceiveP = MAX_SCORE;
        else {
            float score_factor = min(our2inter_time / ball2inter_time, 1.0);
            reverseReceiveP = MAX_SCORE * powf(score_factor, 3);
        }
    }

    return reverseReceiveP;
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_goal(float* me_pos, float* ball_pos, float* their_player_ptr) {
    float reverseGoalP = 0.0;
    float a, b;
    int line_status;
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    line_status = get_line(goal_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            // ���ŵ��з��ľ���
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float dist_goal2opp = dist(goal_pos, their_player_pos);
            if (dist_goal2opp < 60) {
                continue;
            }
            // �жϵз��Ƿ����������Լ��м�
            float r = ((their_player_pos[0] - me_pos[0]) * (goal_pos[0] - me_pos[0]) + (their_player_pos[1] - me_pos[1]) * (goal_pos[1] - me_pos[1])) / (dist2(me_pos, goal_pos));
            if (r < 0 || r > 1) {
                 continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_player_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_player_pos);
            float me2proj_dist = dist(projection_point, me_pos);
            // float goal2proj_dist = dist(projection_point, goal_pos);

            if (opp2proj_dist > 300 || me2proj_dist < 10) { // goal2proj_dist < 10) { // �з����������߽�Զ��з��������ŵ�λ�Ͻ������Բ����ǵз��������ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / me2proj_dist; // goal2proj_dist;
            // ����ʱ���ٽϴ��������㷨ʹ�������е���м��㣬ʵ�����Ż��и���ѡ����˵��˶��Լ����ŵ��赲���С
            if (ratio_dist < 0.3) {
                reverseGoalP += -50 * ratio_dist + 15;
            }
        }
    }
//    // ���ڿ���y=0���߲��׽��������
//    if (abs(me_pos[1]) < 50) {
//        reverseGoalP += 10;
//    }
    return reverseGoalP;
}

__device__ float evaluate_goal_v2(float* me_pos, float* ball_pos_ptr, float* their_player_ptr, int their_goalie) {
    float reverseGoalP = MAX_SCORE;
    float temp_reverseGoalP[3] = { 0.0, 0.0, 0.0 };
    float a, b;
    int line_status;
    float goal_pos[3][2] = { { PITCH_LENGTH / 2, -GOAL_WIDTH / 2 }, { PITCH_LENGTH / 2, 0 }, { PITCH_LENGTH / 2, GOAL_WIDTH / 2 } };
    float ball_max_dist = BALL_MAX_VEL * BALL_MAX_VEL / (2 * BALL_DEC); // �����ɹ�����������
    for (int i = 0; i < 3; i++) {
        // ������Ϊ����ʱ��ס�����õ�λ�����λ��
        float me2goal_dist = dist(goal_pos[i], me_pos);
        float ball_pos_x = me_pos[0] + (ROBOT_FRONT_TO_CENTER + BALL_RADIUS) * (goal_pos[i][0] - me_pos[0]) / me2goal_dist;
        float ball_pos_y = me_pos[1] + (ROBOT_FRONT_TO_CENTER + BALL_RADIUS) * (goal_pos[i][1] - me_pos[1]) / me2goal_dist;
        float ball_pos[2] = { ball_pos_x, ball_pos_y };
        float ball2goal_dist = dist(ball_pos, goal_pos[i]) + BALL_RADIUS + LINE_WIDTH; // ��պý�����Ҫ�����ľ���
        float ball2goal_dir = Normalize(dir(ball_pos, goal_pos[i]));
        if (me2goal_dist > ball_max_dist || fabs(ball2goal_dir) > d2r(LARGEST_GOAL_ANGLE)) { // ̫Զ����ȥ���Ż�Ƕ�̫�󣨿������ߣ��򲻽�ȥ
            temp_reverseGoalP[i] = MAX_SCORE;
            continue;
        }
        // ����ת��ʱ�䣬�Ƕ�Сֱ��touch������turn��Ŀǰ����
        float me2ball_dir = Normalize(dir(me_pos, ball_pos_ptr));
        float turn_dir = fabs(ball2goal_dir - me2ball_dir);
        if (turn_dir > PI) // ת�ӻ��ĽǶ�
            turn_dir = M_2PI - turn_dir;
        float me_turn_time = 0.0;
        if (turn_dir > d2r(LARGEST_TOUCH_ANGLE))
            me_turn_time = TrapezoidalMotionTime(turn_dir, 1);
        line_status = get_line(goal_pos[i], ball_pos, a, b);
        for (int j = 0; j < ENEMY_NUM; j++) {
            if (their_player_ptr[j * POS_INFO_LENGTH]) {
                float* their_player_pos = their_player_ptr + j * POS_INFO_LENGTH + 1;
                // ����ͶӰ��
                float projection_point[2];
                float intercept_point[2];
                get_projection(a, b, line_status, their_player_pos, projection_point);
                // ͶӰ�㲻���������֮��
                if ((projection_point[0] - ball_pos[0]) * (projection_point[0] - goal_pos[i][0]) + (projection_point[1] - ball_pos[1]) * (projection_point[1] - goal_pos[i][1]) > 0) {
                    if (j == their_goalie) { // �з�����Ա�����⴦��
                        intercept_point[0] = goal_pos[i][0];
                        intercept_point[1] = goal_pos[i][1];
                    }
                    else
                        continue;
                }
                else {
                    if (j != their_goalie && is_in_penalty(projection_point[0], projection_point[1], 0)) // ������Ա���ܽ�����
                        continue;
                    else if (j == their_goalie && !is_in_penalty(projection_point[0], projection_point[1], 0)) // ����Ա��������
                        continue;
                    intercept_point[0] = projection_point[0];
                    intercept_point[1] = projection_point[1];
                }
                // ��պù��з�ͶӰ���ʱ��
                float ball2inter_dist = dist(intercept_point, ball_pos) + BALL_RADIUS;
                float ball2inter_vel = sqrt(BALL_MAX_VEL * BALL_MAX_VEL - 2 * BALL_DEC * ball2inter_dist);
                float ball2inter_time = (BALL_MAX_VEL - ball2inter_vel) / BALL_DEC;
                // �з���������պýӴ���ͶӰ���ʱ��
                float opp2inter_dist = dist(intercept_point, their_player_pos) - ROBOT_RADIUS;
                if (opp2inter_dist < 0) // �Ѿ�������������巶Χ�����Խ�ס��
                    opp2inter_dist = 0;
                float opp2inter_time = TrapezoidalMotionTime(opp2inter_dist, 0);

                // ��Ҫ���裬����ʱ��׷�����򣬶��Ǿͽ�����
                float temp_score = 0.0;
                if (opp2inter_time == 0)
                    temp_score = MAX_SCORE;
                else {
                    float score_factor = min((ball2inter_time + me_turn_time) / opp2inter_time, 1.0);
                    temp_score = MAX_SCORE * powf(score_factor, 3);
                }                    
                if (temp_reverseGoalP[i] < temp_score) // ȡ���ķ�ֵ���������ױ����صĵط�
                    temp_reverseGoalP[i] = temp_score;
            }
        }
    }

    // ���ŵ�����������λ�õķ�ֵ����ȡ����С��
    for (int i = 0; i < 3; i++) {
        if (temp_reverseGoalP[i] < reverseGoalP)
            reverseGoalP = temp_reverseGoalP[i];
    }

    return reverseGoalP;
}

__global__ void gpu_calc(float startPos[], float map[])
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // ��������Ĳ���
    int step = startPos[2];
    int me_x = blockIdx.x*step + startPos[0];
    int me_y = threadIdx.x*step + startPos[1];
    float me_pos_ptr[2] = { me_x, me_y };
    float* ball_pos_ptr = startPos + 3;
    float* ball_vel_ptr = startPos + 5;
    float* our_player_ptr = startPos + 7;
    float* their_player_ptr = startPos + (7 + POS_INFO_LENGTH * SELF_NUM);
    // Ϊ������ɫһһ��Ӧ�����ֵ��Ҫ����MAX_SCORE
    if (is_in_penalty(me_x, me_y, ROBOT_RADIUS) || if_collide_theirPlayer(me_pos_ptr, their_player_ptr)) { // ���ܽ����������ܳ�ײ�Է�������
        map[i] = MAX_SCORE;
    }
    else {    
        // ����
        // ��Ҫʹ�õ�һЩ����

        // float me2ball_dist = dist(me_pos_ptr, ball_pos_ptr);
        int their_goalie = check_their_goalie(their_player_ptr);

        // ����ֵ��ԽСԽ��
        // float dist_value = evaluate_dist(me2ball_dist);
        float flat_pass_value_their = evaluate_flat_pass_their(me_pos_ptr, ball_pos_ptr, their_player_ptr, their_goalie);
        float chip_pass_value_their = evaluate_chip_pass_their(me_pos_ptr, ball_pos_ptr, their_player_ptr, their_goalie);
        int our_match_player = rolematch(me_pos_ptr, our_player_ptr);
        float flat_pass_value_our = evaluate_flat_pass_our(me_pos_ptr, ball_pos_ptr, our_player_ptr + our_match_player * POS_INFO_LENGTH + 1);
        float chip_pass_value_our = evaluate_chip_pass_our(me_pos_ptr, ball_pos_ptr, our_player_ptr + our_match_player * POS_INFO_LENGTH + 1);
        float pass_value_their = min(flat_pass_value_their, chip_pass_value_their);
        // ����Է�����Ƚ����ѣ�ֻ�����ҷ��������׳̶ȣ���ʱ������ж�
        float pass_factor_their = min(max(pass_value_their - MAX_SCORE * BEST_TIME_RATIO, 0.0) / (MAX_SCORE * (WOREST_TIME_RATIO - BEST_TIME_RATIO)), 1.0); // 0.7;
        float pass_factor_our = 1 - pass_factor_their; // 0.3;
        float flat_pass_value = flat_pass_value_their * pass_factor_their + flat_pass_value_our * pass_factor_our; // ����ϵ��
        float chip_pass_value = chip_pass_value_their * pass_factor_their + chip_pass_value_our * pass_factor_our;
        // ֻҪ��һ�ִ���ʽ���м���
        float receive_value = min(flat_pass_value, chip_pass_value); // evaluate_receive(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        float goal_value = evaluate_goal_v2(me_pos_ptr, ball_pos_ptr, their_player_ptr, their_goalie); // evaluate_goal(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        // ��ֻ���Ǵ��򣬽�����ֻ�������ţ��м��������Ա������䣬��gpuBestAlgThread����ı߽��Ӧ
        float goal_factor = min(max(ball_pos_ptr[0] + PITCH_LENGTH / 6, 0.0) / (PITCH_LENGTH / 2 - PENALTY_DEPTH + PITCH_LENGTH / 6), 1.0);
        float receive_factor = 1 - goal_factor;
        float final_score = receive_factor * receive_value + goal_factor * goal_value;
        map[i] = min(max(final_score, 0.0), MAX_SCORE);
        // map[i] = me2ball_dist;
    }
}

extern "C" void calc_with_gpu(float* map_cpu, float* start_pos_cpu, int length, int width, int pos_num, float *pitch_info) {
    //clock_t begin, end;
    //begin = clock();
    float* map_gpu, * start_pos_gpu;

    int map_size = length * width * sizeof(float);
    int pos_size = pos_num * sizeof(float);

    hipError_t  status1 = hipMalloc((void**)&map_gpu, map_size); // ���ڴ洢�����Ľ��
    hipError_t  status2 = hipMalloc((void**)&start_pos_gpu, pos_size);
    if (status1 != hipSuccess || status2 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("**************** map calc cuda malloc dbuf error ******************* \r\n");
        return;
    }

    // ��������
    hipError_t status_cpy = hipMemcpy(start_pos_gpu, start_pos_cpu, pos_size, hipMemcpyHostToDevice);
    if (status_cpy != hipSuccess)
    {
        printf("**************** map calc cuda cpy error ******************* \r\n");
        return;
    }

    // ��GPU�п���������Ϣ
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_LENGTH"), pitch_info, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_WIDTH"), pitch_info + 1, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_DEPTH"), pitch_info + 2, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_WIDTH"), pitch_info + 3, sizeof(float));


    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimGrid(length);
    dim3 dimBlock(width);

    // ִ��kernel
    gpu_calc << <dimGrid, dimBlock >> > (start_pos_gpu, map_gpu);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(map_cpu, map_gpu, map_size, hipMemcpyDeviceToHost);

    // �ͷ�GPU�˵��ڴ�
    hipFree(map_gpu);
    hipFree(start_pos_gpu);
    //end = clock();
    //std::cout << "best support point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}


////////////////////////////////////////////////
//       break point calculator from csy      //

// pos info����
// (2+1+2+1+2+2+OURPLAYER_NUM*_player_pos_num+THEIRPLAYER_NUM*_player_pos_num) * sizeof(float)
// ����Ϊ����������Բ�ġ�max_drbble_dist����ǰ��Աλ�á���Ա�������λ�á�����ٶȡ��ҷ�С����λ�á������ٶȣ���λΪ�Ƿ�valid�����з�С����λ�á������ٶȣ���λΪ�Ƿ�valid��
// ����޸��ⲿ�ִ��룬����ϸ�Ķ���ֵ��GPU���ִ��벢��֮������Ӧ���޸�
// ��Ҫע��Ĳ�����CPU�ռ�����룬������и�ֵ�����俽����GPU��ʱ����Ŀռ䡢GPU�Ը��б���Ϣ�Ľ���
__global__ void break_gpu_calc(float pos_info[], float target_info[], float results[], int angle_mod, int dist_mod, float vis_points[])
{
    extern __shared__  float score_map[];

    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    int target_point_idx = blockIdx.x;
    int ang_dist_idx = threadIdx.x;
    int dist_idx = ang_dist_idx % (dist_mod - 1);
    int ang_idx = (ang_dist_idx - dist_idx) / (dist_mod - 1);
    
    // �Ӿ���Ϣ������
    float* dribble_center_point = pos_info;
    float max_dribble_dist = pos_info[2];
    float* self_pos = pos_info + 3;
    float* ball_pos = pos_info + 6;
    float* target_point = target_info + 2 * target_point_idx;
    float* our_player_ptr = pos_info + 10;
    float* their_player_ptr = pos_info + (10 + POS_INFO_LENGTH * SELF_NUM);

    float point_score = 10000;

    //float mod = max_dribble_dist * 2 / (dist_idx + 1);
    float mod = 2 * max_dribble_dist * (dist_idx + 1) / (dist_mod - 1);
    float angle = (ang_idx - angle_mod) * PI / angle_mod + dir(self_pos, target_point);// +(ang_idx - angle_mod) * PI / angle_mod;

    float test_point[2];
    test_point[0] = self_pos[0] + mod * cos(angle);
    test_point[1] = self_pos[1] + mod * sin(angle);
    
    // ʹ��������Բ��
    float vec_dist = dist(test_point, dribble_center_point);
    if (vec_dist > max_dribble_dist) {
        float boundary_point[2];
        boundary_point[0] = (test_point[0] - dribble_center_point[0]) / vec_dist * max_dribble_dist;
        boundary_point[1] = (test_point[1] - dribble_center_point[1]) / vec_dist * max_dribble_dist;
        test_point[0] = dribble_center_point[0] + boundary_point[0];
        test_point[1] = dribble_center_point[1] + boundary_point[1];
    }

    // ���ƶ���������
    MakeInField(test_point);

    // ���ƶ����з�����
    //MakeOutOfTheirPenaltyArea(test_point);

    bool temp_flag = false;
    float a, b;
    int line_status = get_line(self_pos, test_point, a, b);
    for (int i = 0; i < ENEMY_NUM; i++) {
        if (their_player_ptr[i * POS_INFO_LENGTH]) {
            float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
            float test2enemy_dist = dist(their_player_pos, test_point);
            if (test2enemy_dist < 20) {
                temp_flag = true;
                break;
            }
            //// �жϵз��Ƿ���test_point���Լ��м�
            float r = ((their_player_pos[0] - self_pos[0]) * (test_point[0] - self_pos[0]) + (their_player_pos[1] - self_pos[1]) * (test_point[1] - self_pos[1])) / (dist2(self_pos, test_point));
            if (r < 0 || r > 1) {
                continue;
            }
            float projection_point[2];
            get_projection(a, b, line_status, their_player_pos, projection_point);
            if (dist(projection_point, their_player_pos) < 40) {
                temp_flag = true;
                break;
            }
        }
    }

    int cnt = 0;
    if (temp_flag) {
        point_score = 9999;
    }
    else {
        float a1, b1;
        int line_status1 = get_line(test_point, target_point, a1, b1);
        
        // ����ָ��
        float dist_score = dist(test_point, target_point);
        
        float block_score = 8888, near_score = 9999;
        
        for (int i = 0; i < ENEMY_NUM; i++) {
            if (their_player_ptr[i * POS_INFO_LENGTH]) {
                float* their_player_pos = their_player_ptr + i * POS_INFO_LENGTH + 1;
                float straight_dist = dist(their_player_pos, test_point);
                near_score = min(near_score, straight_dist);
                //// �жϵз��Ƿ���test_point��target_point�м�
                float r = ((their_player_pos[0] - target_point[0]) * (test_point[0] - target_point[0]) + (their_player_pos[1] - target_point[1]) * (test_point[1] - target_point[1])) / (dist2(target_point, test_point));
                if (r < 0 || r > 1) {
                    continue;
                }
                cnt++;
                float projection_point[2];
                get_projection(a1, b1, line_status1, their_player_pos, projection_point);
                float projection_dist = dist(projection_point, their_player_pos);
                if (projection_dist > 60) {
                    continue;
                }
                block_score = min(projection_dist, block_score);
            }
        }
        near_score = 1 / (near_score + 1e-8);
        point_score = GPU_COEF_BLOCKSCORE * block_score + GPU_COEF_DISTSCORE * dist_score + GPU_COEF_NEARSCORE * near_score;
    }

    score_map[3 * ang_dist_idx] = point_score;
    score_map[3 * ang_dist_idx + 1] = test_point[0];
    score_map[3 * ang_dist_idx + 2] = test_point[1];
    if (blockIdx.x == 0) {
        vis_points[3 * ang_dist_idx] = point_score;
        vis_points[3 * ang_dist_idx + 1] = test_point[0];
        vis_points[3 * ang_dist_idx + 2] = test_point[1];
    }
    __syncthreads();

    float best_score = 20000;
    int best_idx = 0;
    if (threadIdx.x == 0) {
        for (int i = 0; i < (angle_mod * 2 - 1) * (dist_mod - 1); i++) {
            if (score_map[3 * i] < best_score) {
                best_score = score_map[3 * i];
                best_idx = i;
            }
        }
        results[3 * blockIdx.x] = best_score;
        results[3 * blockIdx.x + 1] = score_map[3 * best_idx + 1];
        results[3 * blockIdx.x + 2] = score_map[3 * best_idx + 2];
    }
    //if (blockIdx.x == 0 && threadIdx.x == 0) {
    //    memcpy(vis_points, score_map, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float));
    //}
}


// target_point_cpu�������ϵĵ�
// target_point_num��Ŀ�����Ŀ
// pos_info_cpu���Ӿ���Ϣ����ͻ����˵�λ��
// pos_info_num���Ӿ���Ϣ��Ŀ

extern "C" int break_calc_with_gpu(float* target_point_cpu, int target_point_num, float* pos_info_cpu, int pos_info_num, int angle_mod, int dist_mod, float* results, float* vis_points_cpu) {
    //clock_t begin, end;
    //begin = clock();
    float* results_gpu, * pos_info_gpu, * target_point_gpu, * vis_points_gpu;

    int result_size = target_point_num * 3 * sizeof(float);
    int pos_size = pos_info_num * sizeof(float);
    int target_info_size = target_point_num * 2 * sizeof(float);

    //std::cout << "pos_info_cpu: " << std::endl;
    //for (int i = 0; i < pos_info_num; i++) {
    //    std::cout << pos_info_cpu[i] << " ";
    //}
    //std::cout << std::endl;
    // ��ȡ�ռ�
    hipError_t  status1 = hipMalloc((void**)&results_gpu, result_size); // ���ڴ洢�����Ľ��
    hipError_t  status2 = hipMalloc((void**)&pos_info_gpu, pos_size);
    hipError_t  status3 = hipMalloc((void**)&target_point_gpu, target_info_size);
    hipError_t  status4 = hipMalloc((void**)&vis_points_gpu, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float));
    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess)
    {
        printf("****************cuda malloc dbuf error ******************* \r\n");
        return 0;
    }

    // ��������
    hipError_t status_cpy1 = hipMemcpy(pos_info_gpu, pos_info_cpu, pos_size, hipMemcpyHostToDevice);
    hipError_t status_cpy2 = hipMemcpy(target_point_gpu, target_point_cpu, target_info_size, hipMemcpyHostToDevice);
    if (status_cpy1 != hipSuccess || status_cpy2 != hipSuccess)
    {
        printf("****************cuda cpy error ******************* \r\n");
        return 0;
    }

    // ��GPU�п���������Ϣ
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_LENGTH"), pitch_info, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_WIDTH"), pitch_info + 1, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_DEPTH"), pitch_info + 2, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_WIDTH"), pitch_info + 3, sizeof(float));


    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimGrid(target_point_num);
    dim3 dimBlock((angle_mod * 2 - 1) * (dist_mod - 1));

    // ִ��kernel
    break_gpu_calc << <dimGrid, dimBlock, 3 * (angle_mod * 2 - 1)* (dist_mod - 1) * sizeof(float) >> > (pos_info_gpu, target_point_gpu, results_gpu, angle_mod, dist_mod, vis_points_gpu); // ����������ָ�������ڴ��С

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(results, results_gpu, result_size, hipMemcpyDeviceToHost);
    hipMemcpy(vis_points_cpu, vis_points_gpu, 3 * (angle_mod * 2 - 1) * (dist_mod - 1) * sizeof(float), hipMemcpyDeviceToHost);
    // �ͷ�GPU�˵��ڴ�
    hipFree(target_point_gpu);
    hipFree(pos_info_gpu);
    hipFree(results_gpu);
    return 1;

    //end = clock();
    //std::cout << "best break point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}


// mat_c = mat_a * mat_b
//�Ż�������cublas��hipblasSgemm���پ�������
// mat_a:M*N, mat_b:N*K, mat_c:M*K
//__global__ void matrix_multi(float* mat_a, float* mat_b, float* mat_c, int M, int N, int K) {
void matrix_multi(float* mat_a, float* mat_b, float* mat_c, int M, int N, int K) {

    hipblasHandle_t handle;
    // initialize CUBLAS context
    hipblasStatus_t stat = hipblasCreate(&handle);

    float al = 1.0f;
    float bet = 0.0f;
    
    //��ǰ�϶�ԭ����洢��ʽΪ������
    stat = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, M, K, N, 
        &al, mat_a, N, mat_b, K, 
        &bet, mat_c, M);
    
    //��Ϊ�����ȿ��������д���
    //stat = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, M, N, K,
    //    &al, mat_a, M, mat_b, K,
    //    &bet, mat_c, M);
    hipblasDestroy(handle);
    //int row_idx = blockIdx.x;
    //int col_idx = threadIdx.x;
    //int idx = blockDim.x * blockIdx.x + threadIdx.x;
    //float c = 0;
    //for (int i = 0; i < N; i++) {
    //    c += mat_a[row_idx * N + i] * mat_b[col_idx * N + i];
    //}
    //mat_c[idx] = c;

}

__global__ void matrix_add(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    MatC[idx] = MatA[idx] + MatB[idx];
}

__global__ void matrix_add_with_relu(float* MatA, float* MatB, float* MatC, int nx, int ny)
{
    int idx = blockDim.x * blockIdx.x + threadIdx.x;
    MatC[idx] = max(MatA[idx] + MatB[idx], 0.0f);
}

//// ��������Ԥ��Ĳ���
//extern "C" int set_ball_model_param(float* a_1_matrix_cpu, float* bias_1_matrix_cpu, float* a_2_matrix_cpu, float* bias_2_matrix_cpu) {
//
//    float* a_1_matrix_gpu, * bias_1_matrix_gpu, * a_2_matrix_gpu, * bias_2_matrix_gpu;
//
//    int result_size = OUTPUT_DIM * sizeof(float);
//    int vel_data_size = INPUT_DIM * sizeof(float);
//    int hidden_layer_size = HIDDEN_LAYER_DIM * sizeof(float);
//
//    hipError_t status1 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_A1_MATRIX), a_1_matrix_cpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float));
//    hipError_t status2 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_BIAS1_MATRIX), bias_1_matrix_cpu, HIDDEN_LAYER_DIM * sizeof(float));
//    hipError_t status3 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_A2_MATRIX), a_2_matrix_cpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float));
//    hipError_t status4 = hipMemcpyToSymbol(HIP_SYMBOL(GPU_BIAS2_MATRIX), bias_2_matrix_cpu, OUTPUT_DIM * sizeof(float));
//
//    //for (int i = 0; i < 10; i++) {
//    //    std::cout << bias_1_matrix_cpu[i] << " ";
//    //}
//    //std::cout << std::endl << std::endl;
//
//    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess || status4 != hipSuccess)
//    {
//        printf("**************** set ball model cuda malloc dbuf error ******************* \r\n");
//        return 0;
//    }
//    return 1;
//}

// ����Ԥ�ⲿ�֣���������һЩGPU�ϵľ�������
extern "C" void ball_model_calc_with_gpu(float* vel_data_cpu, float *predict_results, float* a_1_matrix_cpu, float* bias_1_matrix_cpu, float* a_2_matrix_cpu, float* bias_2_matrix_cpu) {
    //clock_t begin, end;
    //begin = clock();
    float* results_gpu, * vel_data_gpu, * hidden_layer_data_gpu;
    float* a_1_matrix_gpu, * bias_1_matrix_gpu, * a_2_matrix_gpu, * bias_2_matrix_gpu;

    int result_size = OUTPUT_DIM * sizeof(float);
    int vel_data_size = INPUT_DIM * sizeof(float);
    int hidden_layer_size = HIDDEN_LAYER_DIM * sizeof(float);

    // ��ȡ�ռ�
    hipError_t status1 = hipMalloc((void**)&results_gpu, result_size); // ���ڴ洢�����Ľ��
    hipError_t status2 = hipMalloc((void**)&vel_data_gpu, vel_data_size);
    hipError_t status3 = hipMalloc((void**)&hidden_layer_data_gpu, hidden_layer_size);

    hipError_t status4 = hipMalloc((void**)&a_1_matrix_gpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float));
    hipError_t status5 = hipMalloc((void**)&bias_1_matrix_gpu, HIDDEN_LAYER_DIM * sizeof(float));
    hipError_t status6 = hipMalloc((void**)&a_2_matrix_gpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float));
    hipError_t status7 = hipMalloc((void**)&bias_2_matrix_gpu, OUTPUT_DIM * sizeof(float));
    

    if (status1 != hipSuccess || status2 != hipSuccess || status3 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("    Eror text:  %s\n", hipGetErrorString(status1));
        printf("    Eror text:  %s\n", hipGetErrorString(status2));
        printf("    Eror text:  %s\n", hipGetErrorString(status3));
        printf("**************** ball model cuda malloc dbuf error ******************* \r\n");
        return;
    }
    if (status4 != hipSuccess || status5 != hipSuccess || status6 != hipSuccess || status7 != hipSuccess)
    {
        size_t available, total;
        hipMemGetInfo(&available, &total);
        std::cout << "free memory: " << available << " | total memory: " << total << std::endl;
        printf("    Eror text:  %s\n", hipGetErrorString(status4));
        printf("    Eror text:  %s\n", hipGetErrorString(status5));
        printf("    Eror text:  %s\n", hipGetErrorString(status6));
        printf("    Eror text:  %s\n", hipGetErrorString(status7));
        printf("**************** ball model cuda param malloc dbuf error ******************* \r\n");
        return;
    }

    // ��������
    hipError_t status_cpy1 = hipMemcpy(vel_data_gpu, vel_data_cpu, vel_data_size, hipMemcpyHostToDevice);
    hipError_t status_cpy2 = hipMemcpy(a_1_matrix_gpu, a_1_matrix_cpu, INPUT_DIM * HIDDEN_LAYER_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy3 = hipMemcpy(bias_1_matrix_gpu, bias_1_matrix_cpu, HIDDEN_LAYER_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy4 = hipMemcpy(a_2_matrix_gpu, a_2_matrix_cpu, HIDDEN_LAYER_DIM * OUTPUT_DIM * sizeof(float), hipMemcpyHostToDevice);
    hipError_t status_cpy5 = hipMemcpy(bias_2_matrix_gpu, bias_2_matrix_cpu, OUTPUT_DIM * sizeof(float), hipMemcpyHostToDevice);
    if (status_cpy1 != hipSuccess || status_cpy2 != hipSuccess || status_cpy3 != hipSuccess || status_cpy4 != hipSuccess || status_cpy5 != hipSuccess)
    {
        printf("**************** ball model cuda cpy error ******************* \r\n");
        return;
    }

    // y=xA
    //dim3 dimGrid1(1);
    //dim3 dimBlock1(HIDDEN_LAYER_DIM);
    //matrix_multi << <dimGrid1, dimBlock1 >> > (vel_data_gpu, a_1_matrix_gpu, hidden_layer_data_gpu, 1, INPUT_DIM, HIDDEN_LAYER_DIM);
    matrix_multi(vel_data_gpu, a_1_matrix_gpu, hidden_layer_data_gpu, 1, INPUT_DIM, HIDDEN_LAYER_DIM);

    // relu(y+=b)���ӷ�����ʹ��ͬһƬ��ַ�����ǳ˷�����
    dim3 dimGrid2(1);
    dim3 dimBlock2(HIDDEN_LAYER_DIM);
    matrix_add_with_relu << <dimGrid2, dimBlock2 >> > (hidden_layer_data_gpu, bias_1_matrix_gpu, hidden_layer_data_gpu, 1, HIDDEN_LAYER_DIM);

    // y=xA
    //dim3 dimGrid3(1);
    //dim3 dimBlock3(OUTPUT_DIM);
    //matrix_multi << <dimGrid3, dimBlock3 >> > (hidden_layer_data_gpu, a_2_matrix_gpu, results_gpu, 1, HIDDEN_LAYER_DIM, OUTPUT_DIM);
    matrix_multi(hidden_layer_data_gpu, a_2_matrix_gpu, results_gpu, 1, HIDDEN_LAYER_DIM, OUTPUT_DIM);

    // y+=b���ӷ�����ʹ��ͬһƬ��ַ�����ǳ˷�����
    dim3 dimGrid4(1);
    dim3 dimBlock4(OUTPUT_DIM);
    matrix_add << <dimGrid4, dimBlock4 >> > (results_gpu, bias_2_matrix_gpu, results_gpu, 1, OUTPUT_DIM);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(predict_results, results_gpu, result_size, hipMemcpyDeviceToHost);

    // �ͷ�GPU�˵��ڴ�
    hipFree(results_gpu);
    hipFree(vel_data_gpu);
    hipFree(hidden_layer_data_gpu);
    hipFree(a_1_matrix_gpu);
    hipFree(bias_1_matrix_gpu);
    hipFree(a_2_matrix_gpu);
    hipFree(bias_2_matrix_gpu);
    //end = clock();
    //std::cout << "best break point calc time (GPU): " << double(end - begin) / CLOCKS_PER_SEC * 1000 << "ms" << std::endl;
}