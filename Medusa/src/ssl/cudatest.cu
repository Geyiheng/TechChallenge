#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include "hip/hip_runtime.h"
#include "time.h"
#include "param.h"

//namespace {
//    const float PI = 3.1415926;
//    const float M_2PI = PI * 2;
//    const float halfLength = 600;
//    float goal_pos[2] = { halfLength, 0 };
//    const float POINT_PASS_OFF = 200;
//}

__constant__ float PI = 3.1415926;
__constant__ float M_2PI = 6.2831852;
__constant__ float PITCH_WIDTH = 900;
__constant__ float PITCH_LENGTH = 1200;
__constant__ float PENALTY_WIDTH = 240;
__constant__ float PENALTY_DEPTH = 120;
__constant__ float POINT_PASS_OFF = 80.0;
__constant__ int ENEMY_NUM = 8;

extern "C" void get_gpu_info() {
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    for (int i = 0; i < deviceCount; i++)
    {
        hipDeviceProp_t devProp;
        hipGetDeviceProperties(&devProp, i);
//        std::cout << "ʹ��GPU device " << i << ": " << devProp.name << std::endl;
//        std::cout << "�豸ȫ���ڴ������� " << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
//        std::cout << "SM��������" << devProp.multiProcessorCount << std::endl;
//        std::cout << "ÿ���߳̿�Ĺ����ڴ��С��" << devProp.sharedMemPerBlock / 1024.0 << " KB" << std::endl;
//        std::cout << "ÿ���߳̿������߳�����" << devProp.maxThreadsPerBlock << std::endl;
//        std::cout << "�豸��һ���߳̿飨Block���ֿ��õ�32λ�Ĵ��������� " << devProp.regsPerBlock << std::endl;
//        std::cout << "ÿ��EM������߳�����" << devProp.maxThreadsPerMultiProcessor << std::endl;
//        std::cout << "ÿ��EM������߳�������" << devProp.maxThreadsPerMultiProcessor / 32 << std::endl;
//        std::cout << "�豸�϶ദ������������ " << devProp.multiProcessorCount << std::endl;
//        std::cout << "======================================================" << std::endl;
        std::cout << "GPU device " << i << ": " << devProp.name << std::endl;
        std::cout << devProp.totalGlobalMem / 1024 / 1024 << "MB" << std::endl;
    }
}

// �ж�һ�����Ƿ��ڽ���
inline __device__ bool is_in_penalty(float pos_x, float pos_y) {
    if (abs(pos_y) <= PENALTY_WIDTH / 2 && abs(pos_x) >= (PITCH_LENGTH / 2 - PENALTY_DEPTH)) {
        return true;
    }
    else {
        return false;
    }
}

// ���������ľ���
inline __device__ float dist(float* pos_ptr1, float* pos_ptr2) {
    return sqrt((pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]));
}

// ���������ľ���
inline __device__ float dist(float x1, float y1, float * pos_ptr2) {
    return sqrt((x1 - pos_ptr2[0]) * (x1 - pos_ptr2[0]) + (y1 - pos_ptr2[1]) * (y1 - pos_ptr2[1]));
}

// ������������ƽ��
inline __device__ float dist2(float* pos_ptr1, float* pos_ptr2) {
    return (pos_ptr1[0] - pos_ptr2[0]) * (pos_ptr1[0] - pos_ptr2[0]) + (pos_ptr1[1] - pos_ptr2[1]) * (pos_ptr1[1] - pos_ptr2[1]);
}

// �������㹹�������ķ���
inline __device__ float dir(float* pos_ptr1, float* pos_ptr2) {
    return std::atan2(pos_ptr2[1] - pos_ptr1[1], pos_ptr2[0] - pos_ptr1[0]);
}

// ֱ����ֲ��Normalize������ϸ��û�п���
inline __device__ float Normalize(float angle)
{
    // ���ٴֵ���
    angle -= (int)(angle / M_2PI) * M_2PI;
    // ϸ���� (-PI,PI]
    while (angle > PI) {
        angle -= M_2PI;
    }

    while (angle <= -PI) {
        angle += M_2PI;
    }
    return angle;
}

// line_statusΪ1��ʾֱ��б�ʲ�Ϊ������������ʱ��ֱ�߷��� y=ax+b
// line_statusΪ0��ʾֱ��б��Ϊ������������ʱ��ֱ�߷���Ϊx=a
__device__ int get_line(float* line_point1, float* line_point2, float& a, float& b) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        a = line_point1[0];
        b = 0;
        return 0;
    }
    else {
        // ֱ�߷���:y=ax+b
        a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        b = line_point1[1] - a * line_point1[0];
        return 1;
    }
}

// ��һ��������point��������������ֱ���ϵ�������
__device__ float point2line_dist(float *point, float *line_point1, float *line_point2) {
    if (abs(line_point1[0] - line_point2[0]) < 1e-5) {
        return abs(point[0] - line_point1[0]);
    }
    else {
        // ֱ�߷���:y=ax+b
        float a = (line_point1[1] - line_point2[1]) / (line_point1[0] - line_point2[0]);
        float b = line_point1[1] - a * line_point1[0];
        return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
    }
}

// ��һ��������point��������������ֱ�߷��̵�a��b��y=ax+b
__device__ float point2line_dist(float* point, float a, float b) {
    return abs(a * point[0] + b - point[1]) / sqrt(a * a + 1);
}

// ����ͶӰ�㣨���㣩
__device__ void get_projection(float a, float b, int line_status, float* point, float * projection_point) {
    if (line_status) { // ֱ��б��Ϊ����ֵ
        if (abs(a) < 1e-5) { // ֱ����x��ƽ��
            projection_point[0] = point[0];
            projection_point[1] = b;
        }
        else { 
            projection_point[0] = (point[0] / a + point[1] - b) / (a + 1 / a);
            projection_point[1] = a * projection_point[0] + b;
        }
    }
    else { // ֱ��б��Ϊ����
        projection_point[0] = a;
        projection_point[1] = point[1];
    }
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
inline __device__ float evaluate_dist(float dist) {
    if (dist < 200) { // ���뿪���һ������֮��
        return 200 - dist + 100;  // 200 - dist
    }
    else if (dist > 500) {
        return (dist - 500) / 100;  // dist - 500
    }
    else {
        return 0;
    }
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_receive(float *me_pos, float *ball_pos, float * their_player_ptr) {
    float reverseReceiveP = 0.0;

    float a,b;
    int line_status;
    line_status = get_line(ball_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * 6]) {
            // �򵽵з��ľ���
            float* their_palyer_pos = their_player_ptr + i * 6 + 1;
            float dist_ball2opp = dist(ball_pos, their_palyer_pos);
            if (dist_ball2opp < 20) {
                continue;
            }
            //// �жϵз��Ƿ��������Լ��м�
            float r = ((their_palyer_pos[0]-me_pos[0]) * (ball_pos[0] - me_pos[0]) + (their_palyer_pos[1] - me_pos[1]) * (ball_pos[1] - me_pos[1])) / (dist2(me_pos, ball_pos));
            if (r < 0 || r > 1) {
                continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_palyer_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_palyer_pos);
            float ball2proj_dist = dist(projection_point, ball_pos);

            if (opp2proj_dist > 300 || ball2proj_dist < 10) { // �з����봫���߽�Զ��з�������Ͻ������Բ����ǵз��Դ����ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / ball2proj_dist;
            // ����з���Ա�ٶȽ������ٵ�0.6����ñ�ֵ����0.6ʱ�õз���Ա�Դ�����Ӱ��
            // TODO ����ԽԶ�����ֵӦ��Խ��
            if (ratio_dist < 0.6) { 
                reverseReceiveP += -100 * ratio_dist + 60;
            }
        }
    }

    // ��Ҫ�赲�Լ�������
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    float me2ball_dist = dist(me_pos, ball_pos);
    float blockMyself = point2line_dist(me_pos, ball_pos, goal_pos);
    if (blockMyself <= 50 && me2ball_dist < 400 && ball_pos[0] > 100) {
        reverseReceiveP += POINT_PASS_OFF;
    }
    else {
        reverseReceiveP += 20.0 / (blockMyself + 0.1);
    }

    // ���ܹ��ڿ������ߣ����ߴ����׽Ӳ�����
    if (abs(me_pos[1]) > PITCH_WIDTH / 2 - 100) {
        reverseReceiveP += (abs(me_pos[1]) - (PITCH_WIDTH / 2 - 100));
    }

    return reverseReceiveP;
}

/************************************************************************/
/* ���ۺ�������������                                                   */
/************************************************************************/
__device__ float evaluate_goal(float* me_pos, float* ball_pos, float* their_player_ptr) {
    float reverseGoalP = 0.0;
    float a, b;
    int line_status;
    float goal_pos[2] = { PITCH_LENGTH / 2 , 0 };
    line_status = get_line(goal_pos, me_pos, a, b);
    for (int i = 0; i < ENEMY_NUM; i++)
    {
        if (their_player_ptr[i * 6]) {
            // ���ŵ��з��ľ���
            float* their_palyer_pos = their_player_ptr + i * 6 + 1;
            float dist_goal2opp = dist(goal_pos, their_palyer_pos);
            if (dist_goal2opp < 60) {
                continue;
            }
            // �жϵз��Ƿ��������Լ��м�
            float r = ((their_palyer_pos[0] - me_pos[0]) * (ball_pos[0] - me_pos[0]) + (their_palyer_pos[1] - me_pos[1]) * (ball_pos[1] - me_pos[1])) / (dist2(me_pos, goal_pos));
            if (r < 0 || r > 1) {
                continue;
            }
            // ����ͶӰ��
            float projection_point[2];
            get_projection(a, b, line_status, their_palyer_pos, projection_point);
            float opp2proj_dist = dist(projection_point, their_palyer_pos);
            float ball2proj_dist = dist(projection_point, ball_pos);

            if (opp2proj_dist > 300 || ball2proj_dist < 10) { // �з����봫���߽�Զ��з�������Ͻ������Բ����ǵз��Դ����ߵ�Ӱ��
                continue;
            }
            float ratio_dist = opp2proj_dist / ball2proj_dist;
            // ����ʱ���ٽϴ��������㷨ʹ�������е���м��㣬ʵ�����Ż��и���ѡ����˵��˶��Լ����ŵ��赲���С
            if (ratio_dist < 0.3) {
                reverseGoalP += -50 * ratio_dist + 15;
            }
        }
    }
//    // ���ڿ���y=0���߲��׽��������
//    if (abs(me_pos[1]) < 50) {
//        reverseGoalP += 10;
//    }
    return reverseGoalP;
}

__global__ void gpu_calc(float startPos[], float map[])
{
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    // ��������Ĳ���
    int step = startPos[2];
    int me_x = blockIdx.x*step + startPos[0];
    int me_y = threadIdx.x*step + startPos[1];
    // Ϊ������ɫһһ��Ӧ�����ֵ��Ҫ����255
    if (is_in_penalty(me_x, me_y)) {
        map[i] = 255;
    }
    else {
        float me_pos_ptr[2] = { me_x, me_y };
        float* ball_pos_ptr = startPos + 3;
        float* ball_vel_ptr = startPos + 5;
        float* our_player_ptr = startPos + 7;
        float* their_player_ptr = startPos + (7 + 6 * 8);
        // ����
        // ��Ҫʹ�õ�һЩ����

        float me2ball_dist = dist(me_pos_ptr, ball_pos_ptr);

        // ����ֵ
        float dist_value = evaluate_dist(me2ball_dist);
        float receive_value = evaluate_receive(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        float goal_value = evaluate_goal(me_pos_ptr, ball_pos_ptr, their_player_ptr);
        float total_value = dist_value + receive_value + goal_value;
        // float total_value = receive_value;
        // map[i] = me2ball_dist;
        if (total_value < 255) {
            map[i] = total_value;
        }
        else {
            map[i] = 255;
        }
    }
}

extern "C" void calc_with_gpu(float* map_cpu, float* start_pos_cpu, int height, int width, int pos_num, float *pitch_info) {
    float* map_gpu, * start_pos_gpu;

    int map_size = height * width * sizeof(float);
    int pos_size = pos_num * sizeof(float);

    hipError_t  status1 = hipMalloc((void**)&map_gpu, map_size); // ���ڴ洢�����Ľ��
    hipError_t  status2 = hipMalloc((void**)&start_pos_gpu, pos_size);
    if (status1 != hipSuccess || status2 != hipSuccess)
    {
        printf("****************cuda malloc dbuf error ******************* \r\n");
        return;
    }

    // ��������
    hipError_t status_cpy = hipMemcpy(start_pos_gpu, start_pos_cpu, pos_size, hipMemcpyHostToDevice);
    if (status_cpy != hipSuccess)
    {
        printf("****************cuda cpy error ******************* \r\n");
        return;
    }

    // ��GPU�п���������Ϣ
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_LENGTH"), pitch_info, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PITCH_WIDTH"), pitch_info + 1, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_DEPTH"), pitch_info + 2, sizeof(float));
    //hipMemcpyToSymbol(HIP_SYMBOL("PENALTY_WIDTH"), pitch_info + 3, sizeof(float));


    // ����kernelִ�����ã���1024*1024/512����block��ÿ��block������512���߳�
    dim3 dimGrid(height);
    dim3 dimBlock(width);

    // ִ��kernel
    gpu_calc << <dimGrid, dimBlock >> > (start_pos_gpu, map_gpu);

    // ����GPU�˼���õĽ��������CPU��
    hipMemcpy(map_cpu, map_gpu, map_size, hipMemcpyDeviceToHost);

    // �ͷ�GPU�˵��ڴ�
    hipFree(map_gpu);
    hipFree(start_pos_gpu);
}
